
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(const int *a, const int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    const int a = 2, b = 5;
    int c = 0;

    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, sizeof(int));
    hipMalloc((void **)&dev_b, sizeof(int));
    hipMalloc((void **)&dev_c, sizeof(int));

    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);

    vector_add<<<1, 1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d, Is that right?\n", a, b, c);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
