#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <common.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#define block_size 32

using std::swap;


ull get_padding_size(ull size) {
    return size+(block_size-size%block_size)%block_size;
}

void transpose(double *data, ull size) {
    for (ull i = 0; i < size; ++i) {
        for (ull j = i+1; j < size; ++j) {
            swap(data[i*size+j], data[j*size+i]);
        }
    }
}

double *load_padding_mat(const char* file, ull &mat_size, ull &padding_size) {
    FILE *fp = fopen(file, "rb");
    fseek(fp, 0L, SEEK_END);
    ull file_size = ftell(fp);
    rewind(fp);
    mat_size = int_root(file_size/sizeof(double));
    padding_size = get_padding_size(mat_size);
    double *data = (double *)aligned_alloc(64, sizeof(double)*padding_size*padding_size);
    for (ull i = 0; i < padding_size*padding_size; ++i) {
        data[i] = 0;
    }
    if (mat_size == padding_size) {
        fread(data, sizeof(double)*mat_size*mat_size, 1, fp);
    } else {
        for (ull i = 0; i < mat_size; ++i) {
            fread(data+i*padding_size, sizeof(double)*mat_size, 1, fp);
        }
    }
    fclose(fp);
    printf("[info] load a %llux%llu matrix as %llux%llu padding matrix finished\n", mat_size, mat_size, padding_size, padding_size);
    return data;
}

__global__ void matrix_multiply(ui size, double *a, double *b, double *c) {

    __shared__ double sub_a[block_size][block_size];
    __shared__ double sub_b[block_size][block_size];
    ui thread_x = threadIdx.x;
    ui thread_y = threadIdx.y;
    ui block_x = blockIdx.x;
    ui block_y = blockIdx.y;

    ui begin = size*block_size*block_y;
    ui end = begin+size;

    double sum = 0;

    for (ui x = begin; x < end; x += block_size) {
        sub_a[thread_y][thread_x] = a[x+thread_y*size+thread_x];
        sub_b[thread_y][thread_x] = b[x+thread_y*size+thread_x];
        __syncthreads();
#pragma unroll
        for (ui k = 0; k < block_size; ++k) {
            sum += sub_a[thread_y][k]*sub_b[thread_x][k];
        }
        __syncthreads();
    }
    c[size*block_size*block_y+size*thread_y+block_x*block_size+thread_x] = sum;
}

void save_mat(const char* file, double* data, ull mat_size, ull padding_size) {
    FILE *fp = fopen(file, "wb");
    if (mat_size == padding_size) {
        fwrite(data, sizeof(double)*mat_size*mat_size, 1, fp);
    } else {
        for (ull i = 0; i < mat_size; ++i) {
            fwrite(data+i*padding_size, sizeof(double)*mat_size, 1, fp);
        }
    }
    fclose(fp);
}

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("[error] number of arguments is error.\n");
        return 1;
    }

    ull mat_size;
    ull padding_size;

    double *a = load_padding_mat(argv[1], mat_size, padding_size);
    double *b = load_padding_mat(argv[2], mat_size, padding_size);
    transpose(b, padding_size);

    double *device_a;
    double *device_b;
    double *device_c;

    ull mem_size = sizeof(double)*padding_size*padding_size;
    double *c = (double *)aligned_alloc(64, mem_size);

    hipMalloc(&device_a, mem_size);
    hipMalloc(&device_b, mem_size);
    hipMalloc(&device_c, mem_size);

    hipMemcpy(device_a, a, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, mem_size, hipMemcpyHostToDevice);

    dim3 thread_block(block_size, block_size, 1);
    dim3 thread_grid(padding_size/block_size, padding_size/block_size, 1);

    matrix_multiply<<<thread_grid, thread_block>>>(padding_size, device_a, device_b, device_c);
    hipDeviceSynchronize();
    hipMemcpy(c, device_c, mem_size, hipMemcpyDeviceToHost);
    save_mat(argv[3], c, mat_size, padding_size);
    return 0;
}
